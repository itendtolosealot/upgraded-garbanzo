#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipblas.h>
#include <sys/time.h>
#include <hipDNN.h>
#include <syslog.h>
#include "DeepLearning.h"
#include "utils.h"

/* We calculate the sum_exponents for each example in the batch, and use that value to calculate the cross entropy*/
/* sum_exponents is calculated as the sum of the exponents for a given example, i.e., sum_exp = \sum_{i=0}^{output_size} exp(yhat);*/
/* In cross_entropy kernel, the input exp_yhat represents exp(yhat[i]) for all i in {0..., output_size*batch_size} */

__global__ void cross_entropy(int batch_size, int output_size, float* y, float* exp_yhat, float* sum_exponents)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int sum_exp_index = (int) (i / output_size);
  float res = 0;
  if (i < batch_size*output_size) {
	  if ((exp_yhat[i] == 0) || (exp_yhat[i] == 1))
			  res = 0;
		  else
			  res = log(1 - (exp_yhat[i] / sum_exponents[sum_exp_index]))*y[i] + log(exp_yhat[i] /sum_exponents[sum_exp_index])*(1 - y[i]);

	  exp_yhat[i] = res;
  	 }
}

__global__ void vec_sub(float* A, float* B, float* res, int size) {
	  int i = blockIdx.x*blockDim.x + threadIdx.x;
	  if (i < size) {
		  res[i] = A[i] - B[i];
	  }
}

/* We calculate the exponent of the output for every output*/
__global__ void softmax(int array_size, float* out, float* yhat)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	float res = 0;
	if (i < array_size) {
		res = exp(out[i]);
		yhat[i] = res;
	}
}

int setup_descriptors ( struct descriptor** desc, int num_layers, struct layer *layers) {
	struct descriptor* d;
	hipdnnStatus_t status;

	d = (struct descriptor*) malloc(sizeof(descriptor)*num_layers);
	if (d == NULL)
		return 1000;
	for(int i=0;i< num_layers;i++) {
		if(layers[i].type==CONVOLUTION) {
			d[i].valid = true;
			status = hipdnnCreateTensorDescriptor(&d[i].input_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;

			status = hipdnnCreateTensorDescriptor(&d[i].output_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;

			status = hipdnnCreateFilterDescriptor(&d[i].filter_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;

			status = hipdnnCreateConvolutionDescriptor(&d[i].conv_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;

			d[i].d_weights = NULL;

		} else {
			d[i].valid = false;
			status = hipdnnCreateTensorDescriptor(&d[i].y_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnCreateTensorDescriptor(&d[i].output_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnCreateActivationDescriptor(&d[i].acti_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
		}
	}
	*(desc) = d;
	return 0;
}

int destroy_descriptors (struct descriptor* desc, struct cost_descriptor* cost, int num_layers) {
	hipdnnStatus_t status;
	for(int i=0;i< num_layers;i++) {
		if(desc[i].valid) {
			status = hipdnnDestroyTensorDescriptor((desc[i].input_desc));
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnDestroyTensorDescriptor((desc[i].output_desc));
			if(status!= HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnDestroyFilterDescriptor((desc[i].filter_desc));
			if(status!= HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnDestroyConvolutionDescriptor((desc[i].conv_desc));
			if(status!= HIPDNN_STATUS_SUCCESS) return (int)status;

		} else {
			if (desc[i].d_weights != NULL) hipFree(desc[i].d_weights);
			if (desc[i].d_y != NULL) hipFree(desc[i].d_y);
			if (desc[i].d_bias != NULL) hipFree(desc[i].d_bias);
			hipdnnDestroyTensorDescriptor(desc[i].y_desc);
			hipdnnDestroyTensorDescriptor(desc[i].output_desc);
			hipdnnDestroyActivationDescriptor(desc[i].acti_desc);
		}

		if(desc[i].d_input != NULL) hipFree(desc[i].d_input);
		if(desc[i].d_filter != NULL) hipFree(desc[i].d_filter);
		if(desc[i].d_workspace != NULL) hipFree(desc[i].d_workspace);
	}
		if (cost->d_dout != NULL) hipFree(cost->d_dout);
		if (cost->d_out != NULL) hipFree(cost->d_out);
		if (cost->d_one_vec != NULL) hipFree(cost->d_one_vec);
		if (cost->d_y != NULL) hipFree(cost->d_y);
		if (cost->d_yhat != NULL) hipFree(cost->d_yhat);
		if (cost->h_y != NULL) mkl_free(cost->h_y);

	free(desc);
	return 0;
}

int configure_descriptors(hipdnnHandle_t* handle, struct descriptor* desc, int num_layers, struct layer *layers, int batch_size, int IMAGE_WIDTH, int IMAGE_HEIGHT) {
	hipdnnStatus_t status;
	int n,c,h,w;
	for (int i=0; i < num_layers;i++) {
		if (desc[i].valid) {
			if(i==0) {
				status = hipdnnSetTensor4dDescriptor((desc[i].input_desc), HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, IMAGE_HEIGHT, IMAGE_WIDTH);
			} else {
				hipdnnDataType_t t;
				status = hipdnnGetTensor4dDescriptor((desc[i-1].output_desc), &t, &n, &c, &h, &w, NULL, NULL, NULL, NULL);
				status = hipdnnSetTensor4dDescriptor((desc[i].input_desc), HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n,c,h,w);
			}
			int nc = layers[i].conv_layer.num_channels;
			int size = layers[i].conv_layer.filter_size;
			int pad= layers[i].conv_layer.padding;
			int stride = layers[i].conv_layer.stride;
			status = hipdnnSetFilter4dDescriptor((desc[i].filter_desc), HIPDNN_DATA_FLOAT,HIPDNN_TENSOR_NCHW, 1, nc,size,size);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnSetConvolution2dDescriptor((desc[i].conv_desc), pad, pad, stride, stride, 1,1, HIPDNN_CROSS_CORRELATION,HIPDNN_DATA_FLOAT);
			if (status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnGetConvolution2dForwardOutputDim((desc[i].conv_desc), (desc[i].input_desc), (desc[i].filter_desc), &n, &c, &h, &w);
			if (status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnSetTensor4dDescriptor((desc[i].output_desc), HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n,c,h,w);
			if (status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnGetConvolutionForwardAlgorithm(*handle, (desc[i].input_desc), (desc[i].filter_desc),
														(desc[i].conv_desc), (desc[i].output_desc),
														HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,0,
														&desc[i].algo_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnGetConvolutionForwardWorkspaceSize(*handle, (desc[i].input_desc),
															(desc[i].filter_desc), (desc[i].conv_desc),
															(desc[i].output_desc), desc[i].algo_desc,
															&desc[i].workspace_size);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
		} else {
			status = hipdnnSetTensor4dDescriptor((desc[i].output_desc), HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, layers[i].fc_layer.size, 1);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnSetTensor4dDescriptor((desc[i].y_desc), HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, layers[i].fc_layer.size, 1);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnSetActivationDescriptor((desc[i].acti_desc), layers[i].fc_layer.activation, HIPDNN_NOT_PROPAGATE_NAN, 0.5);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
		}
	}
	return 0;
}

hipError_t allocate_memory_cost_desc(struct cost_descriptor* cost, int size_x, int size_y) {
	hipError_t stat;
	stat = hipMalloc(&cost->d_out, size_x*size_y*sizeof(float));
	if (stat != hipSuccess) return stat;
	stat = hipMalloc(&cost->d_dout, size_x*size_y * sizeof(float));
	if (stat != hipSuccess) return stat;
	stat = hipMalloc(&cost->d_yhat, size_x*size_y * sizeof(float));
	if (stat != hipSuccess) return stat;
	stat = hipMalloc(&cost->d_y, size_x*size_y * sizeof(float));
	if (stat != hipSuccess) return stat;
	stat = hipMalloc(&cost->d_one_vec, size_x*size_y * sizeof(float));
	if (stat != hipSuccess) return stat;
	// The sum_exponent will carry the sum of the exponents associated with a particular output (independent of the output size).
	//Hence it would have one value per example. The size_x corresponds to batch_size.
	stat = hipMalloc(&cost->d_sum_exp, size_x*sizeof(float));
	if (stat != hipSuccess) return stat;
	
	cost->h_y = (float*) mkl_malloc(size_x*size_y * sizeof(float), 64);
	if (cost->h_y == NULL) {
		syslog(LOG_ERR, "Unable to allocate memory to h_y");
		return (hipError_t) 2;
	}

	cost->h_one_vec = (float*)mkl_malloc(size_x*size_y* sizeof(float), 64);
	if (cost->h_one_vec == NULL) {
		syslog(LOG_ERR, "Unable to allocate memory to h_y");
		return (hipError_t) 2;
	}
	return hipSuccess;
}

int allocate_memory(struct descriptor* desc, struct cost_descriptor* cost, struct layer* layers, int num_layers, int batch_size, int IMAGE_WIDTH, int IMAGE_HEIGHT) {
	int n,c,h,w;
	hipdnnStatus_t status;
	hipError_t stat;
	hipdnnDataType_t t;
	hipdnnTensorFormat_t format;
	for (int i=0;i<num_layers;i++) {
		if(desc[i].valid) {
			if(i==0) {
				hipMalloc(&desc[i].d_input, batch_size*IMAGE_HEIGHT*IMAGE_WIDTH*sizeof(float));
			} else {
				if (desc[i - 1].valid) {
					status = hipdnnGetTensor4dDescriptor((desc[i - 1].output_desc), &t, &n, &c, &h, &w,
						NULL, NULL, NULL, NULL);
				}
				else {
					n = batch_size;
					c = 1;
					h = 1;
					w = layers[i - 1].fc_layer.size;
				}
				if (status != HIPDNN_STATUS_SUCCESS) { return (int)status; }
				stat = hipMalloc(&desc[i].d_input, n*c*h*w*sizeof(float));
				if (stat != hipSuccess) { return stat; }
			}
			status = hipdnnGetFilter4dDescriptor((desc[i].filter_desc), &t, &format, &n,&c,&h,&w);
			hipMalloc(&desc[i].d_filter, n*c*h*w*sizeof(float));
			if(i==num_layers-1) {
				status = hipdnnGetTensor4dDescriptor((desc[i].output_desc), &t, &n, &c, &h, &w, NULL, NULL, NULL, NULL);
				if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
				stat = allocate_memory_cost_desc(cost, n*c, h*w);
				if (stat != hipSuccess) {
					syslog(LOG_ERR, "Cost struct memory allocation failed with Error %d", stat);
					return stat;
				}
			}
			stat = hipMalloc(&desc[i].d_workspace,desc[i].workspace_size);
			if (stat != hipSuccess) {
				syslog(LOG_ERR, "workspace memory allocation failed with Error %d", stat);
				return stat;
			}

		} else {
				stat = hipMalloc(&desc[i].d_input, layers[i].fc_layer.input_size*sizeof(float));
				if(stat != hipSuccess) return stat;
				stat = hipMalloc(&desc[i].d_weights, (layers[i].fc_layer.input_size)/batch_size*layers[i].fc_layer.size*sizeof(float));
				syslog(LOG_DEBUG, "Memory allocated to d_weights for layer %d PTR to d_weights %p", i, (void *) desc[i].d_weights);
				if(stat != hipSuccess) return stat;
				stat = hipMalloc(&desc[i].d_bias, batch_size*layers[i].fc_layer.size*sizeof(float));
				if(stat != hipSuccess) return stat;
				if(i==num_layers-1) {
					stat = allocate_memory_cost_desc(cost, batch_size, layers[i].fc_layer.size);
					if (stat != hipSuccess) {
						syslog(LOG_ERR, "Cost struct memory allocation failed with Error %d", stat);
						return stat;
					}
				}
				stat = hipMalloc(&desc[i].d_y, (layers[i].fc_layer.size)*batch_size*sizeof(float));
				if(stat != hipSuccess) return stat;

		}


	}
	return 0;
}

int copy_input_to_device(struct descriptor* desc, struct cost_descriptor* cost, struct layer* layers, int num_layers, float* input_image, int batch_size, int IMAGE_WIDTH, int IMAGE_HEIGHT)
{
	hipdnnStatus_t status;
	hipError_t stat;
	hipdnnDataType_t t;
	hipdnnTensorFormat_t format;
	int n,c,h,w;
	FILE* fp = fopen("one_vec.txt","w");

	stat = hipMemcpy(desc[0].d_input, input_image, sizeof(float)*batch_size*IMAGE_WIDTH*IMAGE_HEIGHT, hipMemcpyHostToDevice);
	
	if(stat != hipSuccess) {
		syslog(LOG_ERR, "Encountered Error %d when copying input_image to d_input", stat);
		return stat;
	}

	int size_x;
	int size_y;
	if (desc[num_layers - 1].valid) {
		status = hipdnnGetTensor4dDescriptor((desc[num_layers - 1].output_desc), &t, &n, &c, &h, &w, NULL, NULL, NULL, NULL);
		if (status != 0) {
			syslog(LOG_ERR, "Error while determining Output vec size. Terminating the program");
			return status;
		}
		size_x = n * c;
		size_y = h * w;
	}
	else {
		size_y = layers[num_layers - 1].fc_layer.size;
		size_x = batch_size;
	}

	get_matrix(&cost->h_y, size_x, size_y, 2);
	for(int i = 0; i < size_x*size_y; i++) {
		cost->h_one_vec[i] = 1;
	}

	stat = hipMemcpy(cost->d_one_vec, cost->h_one_vec, size_x*size_y * sizeof(float), hipMemcpyHostToDevice);
	fclose(fp);
	if (stat != hipSuccess) {
		syslog(LOG_ERR, "Error while copying one vector to the device.");
		return stat;
	}
	stat = hipMemcpy(cost->d_y, cost->h_y, size_x*size_y * sizeof(float), hipMemcpyHostToDevice);
	if (stat != hipSuccess) {
		syslog(LOG_ERR, "Error while copying y vector to the device.");
		return stat;
	}

	for(int i=0; i< num_layers; i++) {
		if(desc[i].valid)  {
			status = hipdnnGetFilter4dDescriptor((desc[i].filter_desc), &t, &format, &n,&c,&h,&w);
			if(status != HIPDNN_STATUS_SUCCESS) return status;
			stat = hipMemcpy(desc[i].d_filter, layers[i].conv_layer.filter,
								sizeof(float)*n*c*h*w, hipMemcpyHostToDevice);
			if(stat != hipSuccess) return stat;
		} else {
			stat = hipMemcpy(desc[i].d_weights, layers[i].fc_layer.weights ,
						sizeof(float)*layers[i].fc_layer.input_size*layers[i].fc_layer.size*1/batch_size,
						hipMemcpyHostToDevice);
			if(stat != hipSuccess) return stat;
			for(int j=0; j < batch_size; j++) {
				float* d_bias_matrix = desc[i].d_bias;
				stat = hipMemcpy((d_bias_matrix+j*layers[i].fc_layer.size), layers[i].fc_layer.bias,sizeof(float)*layers[i].fc_layer.size,hipMemcpyHostToDevice);
				if(stat != hipSuccess) return stat;
			}

		}
	}
	return 0;

}


struct Status feedforward(hipdnnHandle_t* cudnn, hipblasHandle_t* handle, struct descriptor* desc, struct cost_descriptor* cost, struct layer *layers, int num_layers, int batch_size)
{
	struct Status ff_stat;
	hipdnnStatus_t status;
	hipblasStatus_t stat;
	hipError_t cuda_stat;
	float* output_array;
	const float alpha=1, beta=0;
//  struct timeval start_timeval, end_timeval;
	ff_stat.cublas_stat = HIPBLAS_STATUS_SUCCESS;
	ff_stat.cuda_stat = hipSuccess;
	ff_stat.cudnn_stat = HIPDNN_STATUS_SUCCESS;

	for(int i=0;i < num_layers;i++) {
        output_array = (i < num_layers-1) ? desc[i+1].d_input:cost->d_out;
		if(desc[i].valid) {
				status = hipdnnConvolutionForward(*cudnn,&alpha, (desc[i].input_desc), desc[i].d_input,
											(desc[i].filter_desc),desc[i].d_filter, (desc[i].conv_desc),
											 desc[i].algo_desc, desc[i].d_workspace,desc[i].workspace_size,
											 &beta, (desc[i].output_desc), output_array);
				if(status != HIPDNN_STATUS_SUCCESS) {
					populate_error_status(&ff_stat, CUDNN, status, i);
					syslog(LOG_ERR, "Convolution Failed at Layer %d with error code: %s", i, hipdnnGetErrorString(status));
					return ff_stat;
				}
		} else {
				assert(desc[i].d_input != NULL);
				assert(desc[i].d_y != NULL );
				assert(desc[i].d_weights != NULL);
				//gettimeofday(&start_timeval, NULL);
				stat =  hipblasSgemm(*handle,
									HIPBLAS_OP_N,
									HIPBLAS_OP_N,
									layers[i].fc_layer.size,
									batch_size,
									(int) ((layers[i].fc_layer.input_size)/batch_size),
									&alpha,
									desc[i].d_weights,
									layers[i].fc_layer.size,
									desc[i].d_input,
									((layers[i].fc_layer.input_size)/batch_size ),
									&beta,
									desc[i].d_y,
									layers[i].fc_layer.size);
				if (stat != HIPBLAS_STATUS_SUCCESS) {
					populate_error_status(&ff_stat, CUBLAS, stat, i);
					syslog(LOG_ERR, "Matrix Multiplication failed with error: %s in layer %d ",cublasGetErrorString(stat),i);
					return ff_stat;
				} else {
					syslog(LOG_DEBUG, "hipblasSgemm routine successful");
				}

				cuda_stat = hipDeviceSynchronize();
				if(cuda_stat != hipSuccess) {
					populate_error_status(&ff_stat, CUDA, cuda_stat, i);
					syslog(LOG_ERR, "hipDeviceSynchronize failed Error code: %s after Matrix Mul in layer %d, Error desc: %s",
							hipGetErrorName(cuda_stat), i, hipGetErrorString(cuda_stat));
					return ff_stat;
				} else {
					syslog(LOG_DEBUG, "cudaDeviceSychronize() after hipblasSgemm successful");
				}

				stat = hipblasSaxpy(*handle, layers[i].fc_layer.size*batch_size, &alpha, desc[i].d_bias, 1, desc[i].d_y, 1);
				if (stat != HIPBLAS_STATUS_SUCCESS) {
					populate_error_status(&ff_stat, CUBLAS, stat, i);
					syslog(LOG_ERR, "Error in hipblasSaxpy routine. Error code: %s", cublasGetErrorString(stat));
					return ff_stat;
				} else {
					syslog(LOG_DEBUG, "hipblasSaxpy routine successful");
				}

				cuda_stat = hipDeviceSynchronize();
				if(cuda_stat != hipSuccess) {
					populate_error_status(&ff_stat, CUDA, cuda_stat, i);
					syslog(LOG_ERR, "hipDeviceSynchronize failed Error code: %s after hipblasSaxpy in layer %d, Error desc: %s",
									hipGetErrorName(cuda_stat), i, hipGetErrorString(cuda_stat));
					return ff_stat;
				} else {
					syslog(LOG_DEBUG, "hipDeviceSynchronize routine successful after hipblasSgemm");
				}

				/* gettimeofday(&end_timeval, NULL);
				float msec_timeval;
				float flop = 2.0*layers[i].fc_layer.size * layers[i].fc_layer.input_size;
				msec_timeval = (end_timeval.tv_sec - start_timeval.tv_sec)*1000.0 + (end_timeval.tv_usec - start_timeval.tv_usec)*1.0/1000.0;
				printf("GFlops using GPU time in MatMul Layer %d Muls: %2.3f is %2.3f\n", i, flop*1e-9 , flop*1.0*1e-6/(msec_timeval));
				gettimeofday(&start_timeval, NULL);*/

				status = hipdnnActivationForward(*cudnn, desc[i].acti_desc, &alpha,
												desc[i].y_desc, desc[i].d_y, &beta,
												desc[i].output_desc , output_array);
				cuda_stat = hipDeviceSynchronize();
				if(cuda_stat != hipSuccess) {
					populate_error_status(&ff_stat, CUDA, cuda_stat, i);
					syslog(LOG_ERR, "hipDeviceSynchronize failed Error code: %s after Activation in layer %d, Error desc: %s",
							hipGetErrorName(cuda_stat), i, hipGetErrorString(cuda_stat));
					return ff_stat;
				} else {
					syslog(LOG_DEBUG, "hipDeviceSynchronize routine successful after Activation");
				}

				/*
				gettimeofday(&end_timeval, NULL);
				flop = 2.0*layers[i].fc_layer.size * batch_size;
				msec_timeval = (end_timeval.tv_sec - start_timeval.tv_sec)*1000.0 + (end_timeval.tv_usec - start_timeval.tv_usec)*1.0/1000.0;
				printf("GfLops using GPU Time at Activation Layer %d  is %2.3f\n", i, flop*1e-6/msec_timeval);
				*/
				if(status != HIPDNN_STATUS_SUCCESS) {
					populate_error_status(&ff_stat, CUDNN, status, i);
					syslog(LOG_ERR, "Activation failed Error code: %s after Activation in layer %d", hipdnnGetErrorString(status), i);
				   return ff_stat;
				} else {
					syslog(LOG_DEBUG, "Activation routine successful");
				}
			}
		}
	ff_stat.failure=NONE;
	return ff_stat;
}

hipError_t compute_gradient_cost_function(struct cost_descriptor* cost, int output_size, int batch_size) {
	int blockSize = 1024;
	int gridSize = ceil(batch_size*output_size/blockSize);
	hipError_t error;
	vec_sub << <gridSize, blockSize >> > (cost->d_yhat, cost->d_y, cost->d_dout, batch_size*output_size);
	error = hipDeviceSynchronize();
	if (error != hipSuccess) {
			syslog(LOG_ERR, "Cost gradient computation failed. Error code: %s, Error Description: %s",
					hipGetErrorName(error), hipGetErrorString(error));
	}
	return error;
}

struct Status feedback(hipdnnHandle_t* cudnn, hipblasHandle_t* handle, struct descriptor* desc, struct cost_descriptor* cost, struct layer *layers, int num_layers, int output_size, int batch_size) {
	struct Status ff_stat;
	hipdnnStatus_t status;
	hipblasStatus_t stat;
	stat = HIPBLAS_STATUS_SUCCESS;
	status= HIPDNN_STATUS_SUCCESS;

	float alpha = -1.0;
	float beta = 0.0;
	ff_stat.cublas_stat = HIPBLAS_STATUS_SUCCESS;
	ff_stat.cuda_stat = hipSuccess;
	ff_stat.cudnn_stat = HIPDNN_STATUS_SUCCESS;

	for (int i = num_layers - 1; i >= 0; i--) {
		float *d_out = (i == num_layers - 1) ? cost->d_out : desc[i + 1].d_input;
		float *d_dout = (i == num_layers - 1) ? cost->d_dout : desc[i + 1].d_din;
		if (desc[i].valid) {
			/* Check for i>0, since inputs are not modified and therefore, there is no need to calculate gradient w.r.t input */
			if (i > 0) {
				status = hipdnnConvolutionBackwardData(*cudnn, &alpha, desc[i].filter_desc, desc[i].d_filter, desc[i].dout_desc, d_dout,
														desc[i].conv_desc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1, desc[i].d_workspace,
														desc[i].workspace_size, &beta, desc[i].din_desc, desc[i].d_din);
				if (status != HIPDNN_STATUS_SUCCESS) {
					syslog(LOG_ERR, "Input gradient computation failed with Error code: %s,  at Layer %d ", hipdnnGetErrorString(status), i);
					populate_error_status(&ff_stat, CUDNN, status, i);
					return ff_stat;
				} else {
					syslog(LOG_DEBUG, "Input gradient computation successful at Layer %d", i);
				}
			}
			status = hipdnnConvolutionBackwardFilter(*cudnn, &alpha, desc[i].input_desc, desc[i].d_input, desc[i].dout_desc, d_dout,
													desc[i].conv_desc, HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1, desc[i].d_workspace,
													desc[i].workspace_size, &beta, desc[i].dfilter_desc, desc[i].d_df);
			if (status != HIPDNN_STATUS_SUCCESS) {
				syslog(LOG_ERR, "Weight gradient computation failed with Error code: %s,  at Layer %d ", hipdnnGetErrorString(status), i);
				populate_error_status(&ff_stat, CUDNN, status, i);
				return ff_stat;
			} else {
				syslog(LOG_DEBUG, "Weight gradient computation successful at Layer %d", i);
			}
		}
		else {
			status = hipdnnActivationBackward(*cudnn, desc[i].acti_desc, &alpha, desc[i].output_desc, d_out,  desc[i].dout_desc,
											 d_dout, desc[i].y_desc, desc[i].d_y, &beta, desc[i].dy_desc, desc[i].d_dy);
			if (status != HIPDNN_STATUS_SUCCESS) {
				syslog(LOG_ERR, "Weight gradient computation failed with Error code: %s,  at Layer %d ", hipdnnGetErrorString(status), i);
				populate_error_status(&ff_stat, CUDNN, status, i);
				return ff_stat;
			} else {
				syslog(LOG_DEBUG, "Weight gradient computation successful at Layer %d", i);
			}
			return ff_stat;
		}
	}
	return ff_stat;
}


int computecost(struct cost_descriptor* cost, int batch_size, int output_size, hipblasHandle_t handle, float* total_cost) {
	hipError_t status;
	hipblasStatus_t stat;
	float alpha = 1;
	float beta = 0;
	int blockSize,gridSize;
	blockSize = 1024;
	gridSize = (int) ceil ((float ) batch_size*output_size/(blockSize));
	//FILE* fp = fopen("d_out.txt","w");
	/* Softmax on every output. The result is stored in yhat itself. */
	softmax << <gridSize, blockSize >> > (batch_size*output_size, cost->d_out, cost->d_yhat);
	status = hipDeviceSynchronize();
	if (status != hipSuccess) {
		syslog(LOG_ERR, "Softmax failed with Error code: %s Error desc: %s", hipGetErrorName(status), hipGetErrorString(status)) ;
		return status;
	}

	/* Matrix mul to find \sum_{i=0}^{output_size} yhat[i]. This will give the sum of exponents for a given exaomple*/
	stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, batch_size, output_size, &alpha, cost->d_one_vec, 1, cost->d_yhat, output_size, &beta, cost->d_sum_exp, 1);
	//print_to_file(fp, cost->d_sum_exp, batch_size, "d_sum_exp", 2, 0);

	if (stat != HIPBLAS_STATUS_SUCCESS) {
		syslog(LOG_ERR, "Softmax sum-exponent failed with Error code: %s", cublasGetErrorString(stat));
		 return status;
	}
	/* Calculating cross entropy knowing the sum of exponents*/
	cross_entropy<<<gridSize, blockSize>>>(batch_size, output_size, cost->d_y, cost->d_yhat, cost->d_sum_exp);
    status = hipDeviceSynchronize();
    if (status != hipSuccess) {
    	syslog(LOG_ERR, "cross_entropy failed with Error code: %s, Error Desc: %s ", hipGetErrorName(status), hipGetErrorString(status));
    	return status;
    }
	//fclose(fp);

	/* Dot product to compute the sum of all the log properties*/
    stat = hipblasSdot(handle, batch_size*output_size, cost->d_one_vec, 1 , cost->d_yhat, 1, total_cost);
    status = hipDeviceSynchronize();
    if (status != hipSuccess) {
    	syslog(LOG_ERR, "Softmax failed with Error code: %s Error desc: %s", hipGetErrorName(status), hipGetErrorString(status)) ;
    	return status;
    }

	if (stat != HIPBLAS_STATUS_SUCCESS) {
		syslog(LOG_ERR, "Total cost computation sum-exponent failed with Error code: %s", cublasGetErrorString(stat));
		return status;
	}
	*total_cost /= (batch_size);
	*total_cost = -(*total_cost);
	return 0;
}
