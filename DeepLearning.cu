#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipblas.h>
#include <sys/time.h>
#include <hipDNN.h>
#include <sys/syslog.h>
#include "DeepLearning.h"

__global__ void cross_entropy(int array_size, float* y, float* yhat)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float res;
  if (i < array_size) {
	  if((yhat[i] == 0)|| (y[i]==0) || (y[i]==1) || (yhat[i] == 1))
		  res = 0;
	  else if (y[i]  > 1)
		  res = __logf(1-yhat[i])*y[i] + __logf(yhat[i])*(1-y[i]);
	  y[i] = res;
  }
}


int setup_descriptors ( struct descriptor** desc, int num_layers, struct layer *layers) {
	struct descriptor* d;
	hipdnnStatus_t status;

	d = (struct descriptor*) malloc(sizeof(struct descriptor)*num_layers);
	if (d == NULL)
		return 1000;
	for(int i=0;i< num_layers;i++) {
		if(layers[i].type==CONVOLUTION) {
			d[i].valid = true;
			status = hipdnnCreateTensorDescriptor(d[i].input_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;

			status = hipdnnCreateTensorDescriptor(d[i].output_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;

			status = hipdnnCreateFilterDescriptor(d[i].filter_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;

			status = hipdnnCreateConvolutionDescriptor(d[i].conv_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;

			d[i].d_weights = NULL;

		} else {
			d[i].valid = false;
			d[i].input_desc= NULL;
			d[i].filter_desc=NULL;
			d[i].output_desc=NULL;
			d[i].conv_desc=NULL;
			status = hipdnnCreateActivationDescriptor(d[i].acti_desc);
		}
	}
	*(desc) = d;
	return 0;
}

int destroy_descriptors (struct descriptor* desc, int num_layers) {
	hipdnnStatus_t status;
	for(int i=0;i< num_layers;i++) {
		if(desc[i].valid) {
			status = hipdnnDestroyTensorDescriptor(*(desc[i].input_desc));
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnDestroyTensorDescriptor(*(desc[i].output_desc));
			if(status!= HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnDestroyFilterDescriptor(*(desc[i].filter_desc));
			if(status!= HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnDestroyConvolutionDescriptor(*(desc[i].conv_desc));
			if(status!= HIPDNN_STATUS_SUCCESS) return (int)status;
		} else {
			hipFree(desc[i].d_weights);
		}
	}
	free(desc);
	return 0;
}

int configure_descriptors(hipdnnHandle_t* handle, struct descriptor* desc, int num_layers, struct layer *layers, int batch_size) {
	hipdnnStatus_t status;
	int n,c,h,w;
	int output_img_width,output_img_height;
	for (int i=0; i < num_layers;i++) {
		if (desc[i].valid) {
			if(i==0) {
				status = hipdnnSetTensor4dDescriptor(*(desc[i].input_desc), HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, 1, IMAGE_HEIGHT, IMAGE_WIDTH);
			} else {
				hipdnnDataType_t t;
				status = hipdnnGetTensor4dDescriptor(*(desc[i-1].output_desc), &t, &n, &c, &h, &w, NULL, NULL, NULL, NULL);
				status = hipdnnSetTensor4dDescriptor(*(desc[i].input_desc), HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n,c,h,w);
			}
			int nc = layers[i].conv_layer.num_channels;
			int size = layers[i].conv_layer.filter_size;
			int pad= layers[i].conv_layer.padding;
			int stride = layers[i].conv_layer.stride;
			int input_img_width = (i==0) ? IMAGE_WIDTH : w;
			int input_img_height = (i==0) ? IMAGE_HEIGHT:h;
			status = hipdnnSetFilter4dDescriptor(*(desc[i].filter_desc), HIPDNN_DATA_FLOAT,HIPDNN_TENSOR_NCHW, 1, nc,size,size);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnSetConvolution2dDescriptor(*(desc[i].conv_desc), pad, pad, stride, stride, 1,1, HIPDNN_CROSS_CORRELATION,HIPDNN_DATA_FLOAT);
			if (status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnGetConvolution2dForwardOutputDim(*(desc[i].conv_desc), *(desc[i].input_desc), *(desc[i].filter_desc), &n, &c, &h, &w);
			if (status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnSetTensor4dDescriptor(*(desc[i].output_desc), HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n,c,h,w);
			if (status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnGetConvolutionForwardAlgorithm(*handle, *(desc[i].input_desc), *(desc[i].filter_desc),
														*(desc[i].conv_desc), *(desc[i].output_desc),
														HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,0,
														&desc[i].algo_desc);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			status = hipdnnGetConvolutionForwardWorkspaceSize(*handle, *(desc[i].input_desc),
															*(desc[i].filter_desc),*(desc[i].conv_desc),
															*(desc[i].output_desc), desc[i].algo_desc,
															&desc[i].workspace_size);
			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
		} else {
			status = hipdnnSetActivationDescriptor(*(desc[i].acti_desc), layers[i].fc_layer.activation,
													HIPDNN_NOT_PROPAGATE_NAN, 0.5);

			if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
		}
	}
	return 0;
}

int allocate_memory(struct descriptor* desc, struct layer* layers, int num_layers, int batch_size) {
	int n,c,h,w;
	hipdnnStatus_t status;
	hipError_t stat;
	hipdnnDataType_t t;
	hipdnnTensorFormat_t format;
	for (int i=0;i<num_layers;i++) {
		if(desc[i].valid) {
			if(i==0) {
				hipMalloc(&desc[i].d_input, batch_size*IMAGE_HEIGHT*IMAGE_WIDTH*sizeof(float));
			} else {
				if (desc[i - 1].valid) {
					status = hipdnnGetTensor4dDescriptor(*(desc[i - 1].output_desc), &t, &n, &c, &h, &w,
						NULL, NULL, NULL, NULL);
				}
				else {
					n = batch_size;
					c = 1;
					h = 1;
					w = layers[i - 1].fc_layer.size;
				}
				if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
				stat = hipMalloc(&desc[i].d_input, n*c*h*w*sizeof(float));
				if(stat != hipSuccess) return stat;
			}
			status = hipdnnGetFilter4dDescriptor(*(desc[i].filter_desc), &t, &format, &n,&c,&h,&w);
			hipMalloc(&desc[i].d_filter, n*c*h*w*sizeof(float));
			if(i==num_layers-1) {
				status = hipdnnGetTensor4dDescriptor(*(desc[i].output_desc), &t, &n, &c, &h, &w,
													NULL, NULL, NULL, NULL);
				if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
				stat = hipMalloc(&desc[i].d_output,n*c*h*w*sizeof(float));
				if(stat != hipSuccess) return stat;
			}
			stat = hipMalloc(&desc[i].d_workspace,desc[i].workspace_size);
			if(stat != hipSuccess) return stat;

		} else {
				stat = hipMalloc(&desc[i].d_input, layers[i].fc_layer.input_size*sizeof(float));
				if(stat != hipSuccess) return stat;
				stat = hipMalloc(&desc[i].d_weights,
							(layers[i].fc_layer.input_size)/batch_size*layers[i].fc_layer.size*sizeof(float));
				if(stat != hipSuccess) return stat;
				if(i==num_layers-1) {
					stat = hipMalloc(&desc[i].d_output, batch_size*layers[i].fc_layer.size*sizeof(float));
					if(stat != hipSuccess) return stat;
				}
		}


	}
	return 0;
}

int copy_input_to_device(struct descriptor* desc, struct layer* layers, int num_layers, float* input_image, int batch_size)
{
	hipdnnStatus_t status;
	hipError_t stat;
	hipdnnDataType_t t;
	hipdnnTensorFormat_t format;
	int n,c,h,w;

	stat = hipMemcpy(desc[0].d_input, input_image, sizeof(float)*batch_size*IMAGE_WIDTH*IMAGE_HEIGHT, hipMemcpyHostToDevice);
	for(int i=0; i< num_layers; i++) {
		if(desc[i].valid)  {
			status = hipdnnGetFilter4dDescriptor(*(desc[i].filter_desc), &t, &format, &n,&c,&h,&w);
			if(status != HIPDNN_STATUS_SUCCESS) return stat;
			stat = hipMemcpy(desc[i].d_filter, layers[i].conv_layer.filter,
								sizeof(float)*n*c*h*w, hipMemcpyHostToDevice);
			if(stat != hipSuccess) return stat;
		} else {
			stat = hipMemcpy(desc[i].d_weights, layers[i].fc_layer.weights ,
						sizeof(float)*layers[i].fc_layer.input_size*layers[i].fc_layer.size,
						hipMemcpyHostToDevice);
			if(stat != hipSuccess) return stat;
		}
	}
	return 0;

}


int feedforward(hipdnnHandle_t* cudnn, 	hipblasHandle_t* handle, struct descriptor* desc, struct layer *layers, int num_layers, int batch_size)
{
	hipdnnStatus_t status;
	hipblasStatus_t stat;
	float* output_array;
	const float alpha=1, beta=0;
	for(int i=0;i < num_layers;i++) {
        output_array = (i < num_layers-1) ? desc[i+1].d_input:desc[i].d_output;
		if(desc[i].valid) {
				status = hipdnnConvolutionForward(*cudnn,&alpha, *(desc[i].input_desc), desc[i].d_input,
											*(desc[i].filter_desc),desc[i].d_filter, *(desc[i].conv_desc),
											 desc[i].algo_desc, desc[i].d_workspace,desc[i].workspace_size,
											 &beta, *(desc[i].output_desc), output_array);
				if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
		} else {
				stat =  hipblasSgemm(*handle,
									HIPBLAS_OP_N,
									HIPBLAS_OP_N,
									layers[i].fc_layer.size,
									batch_size,
									(int) ((layers[i].fc_layer.input_size)/batch_size),
									&alpha,
									desc[i].d_weights,
									layers[i].fc_layer.size,
									desc[i].d_input,
									(int)((layers[i].fc_layer.input_size)/batch_size ),
									&beta,
									output_array,
									layers[i].fc_layer.size);
				if (stat != HIPBLAS_STATUS_SUCCESS) return (int) stat;

				status = hipdnnActivationForward(*cudnn,
												*(desc[i].acti_desc),
												&alpha,
												*(desc[i].output_desc),
												(void *)output_array,
												&beta,
												*(desc[i].output_desc),
												(void *) output_array);
				if(status != HIPDNN_STATUS_SUCCESS) return (int)status;
			}
		}
	return 0;
}


int computecost(float* y, float* yhat, float* ones_vector, int size, hipblasHandle_t* handle, float* cost) {
	hipError_t status;
	hipblasStatus_t stat;
	cross_entropy<<<(size+255)/256, 256>>>(size, y, yhat);
    status = hipDeviceSynchronize();
    if (status != hipSuccess) return (int) status;
    stat = hipblasSdot(*handle, size, ones_vector,1, y, 1, cost);
	if (stat != HIPBLAS_STATUS_SUCCESS) return (int) stat;
	return 0;
}